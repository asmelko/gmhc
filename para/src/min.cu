#include "hip/hip_runtime.h"
#include <kernels.cuh>
#include <clustering.hpp>
#include <iostream>
#include <cstdio>

#include <>

void cuda_check(hipError_t code, const char* file, int line)
{
    if (code != hipSuccess)
    {
        std::cerr << hipGetErrorString(code) << " at " << file << ":" << line;
        exit(code);
    }
}

void cuBLAS_check(hipblasStatus_t code, const char* file, int line)
{
    if (code != HIPBLAS_STATUS_SUCCESS)
    {
        switch (code)
        {
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            std::cerr << "HIPBLAS_STATUS_NOT_INITIALIZED" << " at " << file << ":" << line;
            return;

        case HIPBLAS_STATUS_ALLOC_FAILED:
            std::cerr << "HIPBLAS_STATUS_ALLOC_FAILED" << " at " << file << ":" << line;
            return;

        case HIPBLAS_STATUS_INVALID_VALUE:
            std::cerr << "HIPBLAS_STATUS_INVALID_VALUE" << " at " << file << ":" << line;
            return;

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            std::cerr << "HIPBLAS_STATUS_ARCH_MISMATCH" << " at " << file << ":" << line;
            return;

        case HIPBLAS_STATUS_MAPPING_ERROR:
            std::cerr << "HIPBLAS_STATUS_MAPPING_ERROR" << " at " << file << ":" << line;
            return;

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            std::cerr << "HIPBLAS_STATUS_EXECUTION_FAILED" << " at " << file << ":" << line;
            return;

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            std::cerr << "HIPBLAS_STATUS_INTERNAL_ERROR" << " at " << file << ":" << line;
            return;
        }
        std::cerr << "Unknown cuBLAS error" << " at " << file << ":" << line;
    }
}

__device__ void print_point(const float* data, size_t x, size_t dim)
{
    for (size_t i = 0; i < dim; i++)
    {
        printf("%f ", data[x * dim + i]);
    }
    printf("\n");
}

__device__ void print_min(const clustering::chunk_t* output)
{
    printf("%f %d %d\n", output->min_dist, output->min_i,output->min_j);
}

__inline__ __device__ float euclidean_norm(const float* mem, size_t x, size_t y, size_t dim)
{
    float tmp_sum = 0;
    for (size_t i = 0; i < dim; ++i)
    {
        auto tmp = mem[x * dim + i] - mem[y * dim + i];
        tmp_sum += tmp * tmp;
    }
    return sqrtf(tmp_sum);
}

__inline__ __device__ size2 load_data(const float* points, float const * const * inverses, size_t count, size_t dim, float* dest, size_t hsize, size2 coords)
{
    size_t up_point_offset = hsize * coords.x;
    size_t left_point_offset = hsize * coords.y;

    auto up_ptr = points + up_point_offset * dim;
    auto left_ptr = points + left_point_offset * dim;

    size_t up_size = up_point_offset + hsize > count ? count - up_point_offset : hsize;
    size_t left_size = left_point_offset + hsize > count ? count - left_point_offset : hsize;

    for (size_t i = threadIdx.x; i < up_size + left_size; i += blockDim.x)
    {
        if (i < up_size)
            memcpy(dest + i * dim, up_ptr + i * dim, dim * sizeof(float));
        else
            memcpy(dest + i * dim, left_ptr + (i - up_size) * dim, dim * sizeof(float));
    }

    const float** inv_dest = reinterpret_cast<const float**>(dest + (up_size + left_size) * dim);

    for (size_t i = threadIdx.x; i < up_size + left_size; i += blockDim.x)
    {
        if (i < up_size)
            inv_dest[i] = inverses[hsize * coords.x + i];
        else
            inv_dest[i] = inverses[hsize * coords.y + (i - up_size)];
    }

    return { up_size, left_size };
}

__inline__ __device__ clustering::chunk_t reduce_min_warp(clustering::chunk_t data)
{
    for (unsigned int offset = warpSize / 2; offset > 0; offset /= 2)
    {
        auto tmp_dist = __shfl_down_sync(0xFFFFFFFF, data.min_dist, offset);
        auto tmp_i = __shfl_down_sync(0xFFFFFFFF, data.min_i, offset);
        auto tmp_j = __shfl_down_sync(0xFFFFFFFF, data.min_j, offset);
        if (tmp_dist < data.min_dist)
        {
            data.min_dist = tmp_dist;
            data.min_i = tmp_i;
            data.min_j = tmp_j;
        }
    }
    return data;
}

__inline__ __device__ clustering::chunk_t reduce_min_block(clustering::chunk_t data, clustering::chunk_t* shared_mem)
{
    data = reduce_min_warp(data);

    auto lane_id = threadIdx.x % warpSize;
    auto warp_id = threadIdx.x / warpSize;

    if (lane_id == 0)
        shared_mem[warp_id] = data;

    __syncthreads();

    data = (threadIdx.x < blockDim.x / warpSize) ? shared_mem[threadIdx.x] : shared_mem[0];

    data = reduce_min_warp(data);
    return data;
}

__global__ void reduce_min(const clustering::chunk_t* input, clustering::chunk_t* output, size_t input_size)
{
    static __shared__ clustering::chunk_t shared_mem[32];

    clustering::chunk_t min;
    min.min_dist = FLT_MAX;

    for (size_t i = threadIdx.x; i < input_size; i += blockDim.x)
    {
        auto tmp = input[i];
        if (tmp.min_dist < min.min_dist)
            min = tmp;
    }

    min = reduce_min_block(min, shared_mem);

    if (threadIdx.x == 0)
        *output = min;
}

__inline__ __device__ size2 compute_coordinates(size_t count_in_line, size_t plain_index)
{  
    size_t y = 0;
    while (plain_index >= count_in_line)
    {
        y++;
        plain_index -= count_in_line--;
    }
    return { plain_index + y, y };
}

__inline__ __device__ clustering::chunk_t diagonal_loop(size_t block_size, size_t dim, float* shared_mem)
{
    clustering::chunk_t min;
    min.min_dist = FLT_MAX;

    for (size_t i = threadIdx.x; i < (((block_size + 1) * block_size) / 2) - block_size; i += blockDim.x)
    {
        auto coords = compute_coordinates(block_size - 1, i);
        coords.x++;

        float dist;
        float** inv_mem = reinterpret_cast<float**>(shared_mem + (block_size * 2)*dim);
        if (!inv_mem[coords.x] && !inv_mem[coords.y])
            dist = euclidean_norm(shared_mem, coords.x, coords.y + block_size, dim);

        if (min.min_dist > dist)
        {
            min.min_dist = dist;
            min.min_i = coords.y;
            min.min_j = coords.x;
        }
    }
    return min;
}

__inline__ __device__ clustering::chunk_t non_diagonal_loop(size2 chunk_dim, size_t dim, float* shared_mem)
{
    clustering::chunk_t min;
    min.min_dist = FLT_MAX;

    for (size_t i = threadIdx.x; i < chunk_dim.x * chunk_dim.y; i += blockDim.x)
    {
        auto x = i % chunk_dim.x;
        auto y = i / chunk_dim.x;

        float dist;
        float** inv_mem = reinterpret_cast<float**>(shared_mem + (chunk_dim.x + chunk_dim.y)*dim);
        if (!inv_mem[x] && !inv_mem[y])
            dist = euclidean_norm(shared_mem, x, y + chunk_dim.x, dim);

        if (min.min_dist > dist)
        {
            min.min_dist = dist;
            min.min_i = y;
            min.min_j = x;
        }
    }
    return min;
}

__inline__ __device__ clustering::chunk_t block_euclidean_min(const float* points, size_t count, size_t dim, float* shared_mem, size_t hshsize, size2 coords, const float* const* inverses)
{
    auto sh_sizes = load_data(points, inverses, count, dim, shared_mem, hshsize, coords);

    __syncthreads();

    clustering::chunk_t min;

    if (coords.x == coords.y)
        min = diagonal_loop(sh_sizes.x, dim, shared_mem);
    else
        min = non_diagonal_loop(sh_sizes, dim, shared_mem);

    min.min_i += coords.y * hshsize;
    min.min_j += coords.x * hshsize;

    clustering::chunk_t* tmp_res = reinterpret_cast<clustering::chunk_t*>(shared_mem);

    min = reduce_min_block(min, tmp_res);

    return min;
}

__global__ void euclidean_min(const float* points, size_t point_count, size_t point_dim, size_t half_shared_size, clustering::chunk_t* res, size_t chunks_in_line, size_t chunk_count, const float* const* inverses)
{
    extern __shared__ float shared_mem[];

    for (size_t i = blockIdx.x; i < chunk_count; i += gridDim.x)
    {
        auto coords = compute_coordinates(chunks_in_line, i);

        auto block_min = block_euclidean_min(points, point_count, point_dim, shared_mem, half_shared_size, coords, inverses);

        if (threadIdx.x == 0)
            res[i] = block_min;
    }
}

void run_euclidean_min(const input_t in, clustering::chunk_t* out, const float * const * inverses, kernel_info info)
{
    auto half_shared_size = info.shared_size / 2;
    auto chunks_in_line = (in.count + half_shared_size - 1) / half_shared_size;
    auto chunk_count =  ((chunks_in_line + 1) * chunks_in_line) / 2;

    euclidean_min << <info.grid_dim, info.block_dim, info.shared_size * in.dim * sizeof(float) + info.shared_size * sizeof(float*) >> > (in.data, in.count, in.dim, half_shared_size, out, chunks_in_line, chunk_count, inverses);
    reduce_min << <1, 1024 >> > (out, out, chunk_count);
}

void run_min(const input_t in, clustering::chunk_t* out, const float* const* inverses, kernel_info info)
{
    auto half_shared_size = info.shared_size / 2;
    auto chunks_in_line = (in.count + half_shared_size - 1) / half_shared_size;
    auto chunk_count = ((chunks_in_line + 1) * chunks_in_line) / 2;

    euclidean_min << <info.grid_dim, info.block_dim, info.shared_size* in.dim * sizeof(float) >> > (in.data, in.count, in.dim, half_shared_size, out, chunks_in_line, chunk_count, inverses);
}

clustering::chunk_t run_reduce(const clustering::chunk_t* chunks, clustering::chunk_t* out, size_t chunk_count, kernel_info info)
{
    reduce_min << <1, 1024 >> > (chunks, out, chunk_count);
    CUCH(hipDeviceSynchronize());
    clustering::chunk_t res;
    CUCH(hipMemcpy(&res, out, sizeof(clustering::chunk_t), hipMemcpyKind::hipMemcpyDeviceToHost));
    return res;
}