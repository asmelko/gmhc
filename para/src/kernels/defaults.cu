#include "hip/hip_runtime.h"
#include <kernels.cuh>
#include <>
#include <cfloat>

using namespace clustering;

__global__ void set_default_asgn(asgn_t* __restrict__ asgns, csize_t size)
{
	for (csize_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += gridDim.x * blockDim.x)
		asgns[i] = i;
}

void run_set_default_asgn(asgn_t* asgns, csize_t N)
{
	set_default_asgn << <50, 1024 >> > (asgns, N);
}

__global__ void set_default(float* __restrict__ icov_matrix, csize_t size)
{
	for (csize_t i = threadIdx.x; i < size; i += blockDim.x)
		icov_matrix[i] = 1;
}

void run_set_default_inverse(float* icov_matrix, csize_t size)
{
	set_default << <1, size >> > (icov_matrix, size);
}

__global__ void set_default_neigh(neighbour_t* neighbours, csize_t count)
{
	for (csize_t i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += gridDim.x * blockDim.x)
		neighbours[i].distance = FLT_MAX;
}

void run_set_default_neigh(neighbour_t* neighbours, csize_t count, kernel_info info)
{
	set_default_neigh<< <info.grid_dim, info.block_dim>> > (neighbours, count);
}