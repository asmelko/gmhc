#include "hip/hip_runtime.h"
#include <kernels.cuh>
#include <>

using namespace clustering;

__global__ void set_default_asgn(asgn_t* __restrict__ asgns, csize_t size)
{
	for (csize_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += gridDim.x * blockDim.x)
		asgns[i] = i;
}

void run_set_default_asgn(asgn_t* asgns, csize_t N)
{
	set_default_asgn << <50, 1024 >> > (asgns, N);
}

__global__ void set_default(float* __restrict__ icov_matrix, csize_t size)
{
	for (csize_t i = threadIdx.x; i < size; i += blockDim.x)
		icov_matrix[i] = 1;
}

void run_set_default_inverse(float* icov_matrix, csize_t size)
{
	set_default << <1, size >> > (icov_matrix, size);
}
