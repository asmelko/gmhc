#include "hip/hip_runtime.h"
#include <>

#include <cub/block/block_reduce.cuh>

#include "common_kernels.cuh"
#include "kernels.cuh"

using namespace clustering;

__constant__ float expected_point[MAX_DIM];

void assign_constant_storage(const float* value, csize_t size, hipMemcpyKind kind, hipStream_t stream)
{
    CUCH(hipMemcpyToSymbolAsync(HIP_SYMBOL(expected_point), value, size, (size_t)0, kind, stream));
}

#define BUFF_SIZE 32

template<size_t DIM_X>
__global__ void covariance(const float* __restrict__ points,
    float* __restrict__ cov_matrix,
    csize_t count,
    csize_t dim)
{
    typedef hipcub::BlockReduce<float, DIM_X> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    float cov_point[BUFF_SIZE];

    csize_t cov_idx = 0;
    csize_t cov_size = ((dim + 1) * dim) / 2;

    while (cov_idx < cov_size)
    {
        auto need = cov_size - cov_idx;
        need = need > BUFF_SIZE ? BUFF_SIZE : need;
        auto end = cov_idx + need;

        memset(cov_point, 0, need * sizeof(float));

        for (csize_t idx = blockDim.x * blockIdx.x + threadIdx.x; idx < count; idx += gridDim.x * blockDim.x)
        {
            for (csize_t point_idx = cov_idx; point_idx < end; point_idx++)
            {
                auto coords = compute_coordinates(dim, point_idx);
                cov_point[point_idx - cov_idx] += (points[idx * dim + coords.x] - expected_point[coords.x])
                    * (points[idx * dim + coords.y] - expected_point[coords.y]);
            }
        }

        for (csize_t i = 0; i < need; i++)
        {
            float aggregate = BlockReduce(temp_storage).Sum(cov_point[i]);

            if (threadIdx.x == 0)
                cov_matrix[blockIdx.x * cov_size + cov_idx + i] = aggregate;
        }

        cov_idx += need;
    }
}

__global__ void finish_covariance(const float* __restrict__ in_cov_matrix,
    float* __restrict__ out_cov_matrix,
    csize_t grid_size,
    csize_t divisor,
    csize_t dim)
{
    csize_t cov_size = ((dim + 1) * dim) / 2;

    for (csize_t i = threadIdx.x; i < cov_size; i += blockDim.x)
    {
        float sum = 0;
        for (size_t j = 0; j < grid_size; j++)
        {
            sum += in_cov_matrix[j * cov_size + i];
        }
        sum /= divisor;

        auto coords = compute_coordinates(dim, i);
        out_cov_matrix[coords.x + coords.y * dim] = sum;
        out_cov_matrix[coords.x * dim + coords.y] = sum;
    }
}

__global__ void store_icov_data(float* __restrict__ icov_dest,
    float* __restrict__ mf_dest,
    const float* __restrict__ icov_src,
    const float mf_src,
    clustering::csize_t dim)
{
    csize_t cov_size = ((dim + 1) * dim) / 2;

    for (csize_t idx = threadIdx.x; idx < cov_size; idx += blockDim.x)
    {
        auto coords = compute_coordinates(dim, idx);

        if (coords.x == coords.y)
            icov_dest[idx] = icov_src[coords.x + coords.y * dim];
        else
            icov_dest[idx] = 2 * icov_src[coords.x + coords.y * dim];
    }

    if (threadIdx.x == 0 && mf_dest)
        *mf_dest = mf_src;
}

__device__ void reduce_mul_warp(float* __restrict__ point)
{
    for (unsigned int offset = warpSize / 2; offset > 0; offset /= 2)
        *point *= __shfl_down_sync(0xFFFFFFFF, *point, offset);
}

__device__ void reduce_mul_block(float* __restrict__ point, float* __restrict__ shared_mem)
{
    reduce_mul_warp(point);

    auto lane_id = threadIdx.x % warpSize;
    auto warp_id = threadIdx.x / warpSize;

    if (lane_id == 0)
        memcpy(shared_mem + warp_id, point, sizeof(float));

    __syncthreads();

    *point = (threadIdx.x < blockDim.x / warpSize) ? shared_mem[threadIdx.x] : 1;

    reduce_mul_warp(point);
}

__global__ void transform_cov(float* __restrict__ matrix,
    csize_t dim,
    float weight_factor,
    bool use_cholesky,
    const float* __restrict__ cholesky_decomp,
    const int* __restrict__ cholesky_success)
{
    __shared__ float shared[1];

    float mf = 1.f;

    if (use_cholesky && *cholesky_success == 0)
    {
        for (csize_t idx = threadIdx.x; idx < dim; idx += blockDim.x)
            mf *= powf(cholesky_decomp[idx * (dim + 1)], 2.f / dim);

        __syncthreads();

        reduce_mul_warp(&mf);

        if (threadIdx.x == 0)
            shared[0] = mf;

        __syncthreads();

        mf = shared[0];
    }

    csize_t cov_size = ((dim + 1) * dim) / 2;

    for (csize_t idx = threadIdx.x; idx < cov_size; idx += blockDim.x)
    {
        auto coords = compute_coordinates(dim, idx);

        matrix[coords.x + coords.y * dim] =
            weight_factor * matrix[coords.x + coords.y * dim] + (1 - weight_factor) * mf * (coords.x == coords.y);
    }
}

__global__ void compute_store_icov_mf(float* __restrict__ dest, csize_t dim, const float* __restrict__ cholesky_decomp)
{
    float icmf = 1.f;

    for (csize_t idx = threadIdx.x; idx < dim; idx += blockDim.x)
        icmf *= powf(cholesky_decomp[idx * (dim + 1)], -2.f / dim);

    __syncthreads();

    reduce_mul_warp(&icmf);

    if (threadIdx.x == 0)
        *dest = icmf;
}


void run_covariance(const float* points,
    float* work_covariance,
    float* out_covariance,
    csize_t cluster_size,
    csize_t dim,
    kernel_info info)
{
    auto block_dim = ((cluster_size + 31) / 32) * 32;
    auto grid_dim = (block_dim + 1023) / 1024;
    block_dim = block_dim > info.block_dim ? info.block_dim : block_dim;
    grid_dim = grid_dim > info.grid_dim ? info.grid_dim : grid_dim;

    if (block_dim == 32)
        covariance<32><<<grid_dim, 32, 0, info.stream>>>(points, work_covariance, cluster_size, dim);
    else if (block_dim <= 64)
        covariance<64><<<grid_dim, 64, 0, info.stream>>>(points, work_covariance, cluster_size, dim);
    else if (block_dim <= 128)
        covariance<128><<<grid_dim, 128, 0, info.stream>>>(points, work_covariance, cluster_size, dim);
    else if (block_dim <= 256)
        covariance<256><<<grid_dim, 256, 0, info.stream>>>(points, work_covariance, cluster_size, dim);
    else if (block_dim <= 512)
        covariance<512><<<grid_dim, 512, 0, info.stream>>>(points, work_covariance, cluster_size, dim);
    else
        covariance<1024><<<grid_dim, 1024, 0, info.stream>>>(points, work_covariance, cluster_size, dim);

    finish_covariance<<<1, 32, 0, info.stream>>>(work_covariance, out_covariance, grid_dim, cluster_size, dim);
}

void run_store_icovariance_data(float* icov_dest,
    float* mf_dest,
    const float* icov_src,
    const float mf_src,
    clustering::csize_t dim,
    hipStream_t stream)
{
    store_icov_data<<<1, 32, 0, stream>>>(icov_dest, mf_dest, icov_src, mf_src, dim);
}

void run_transform_cov(float* matrix,
    csize_t dim,
    float weight_factor,
    bool use_cholesky,
    const float* cholesky_decomp,
    const int* cholesky_success,
    hipStream_t stream)
{
    transform_cov<<<1, 32, 0, stream>>>(matrix, dim, weight_factor, use_cholesky, cholesky_decomp, cholesky_success);
}

void run_compute_store_icov_mf(float* dest, csize_t dim, const float* cholesky_decomp, hipStream_t stream)
{
    compute_store_icov_mf<<<1, 32, 0, stream>>>(dest, dim, cholesky_decomp);
}