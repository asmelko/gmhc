#include "hip/hip_runtime.h"
#include <>

#include "common_kernels.cuh"
#include "kernels.cuh"

using namespace clustering;

__constant__ float expected_point[MAX_DIM];

void assign_constant_storage(const float* value, csize_t size, hipMemcpyKind kind)
{
    CUCH(hipMemcpyToSymbol(HIP_SYMBOL(expected_point), value, size, (size_t)0, kind));
}

__inline__ __device__ void reduce_sum_warp(float* __restrict__ cov, csize_t size, unsigned mask)
{
    for (csize_t i = 0; i < size; ++i)
    {
        float tmp = cov[i];
        for (unsigned int offset = warpSize / 2; offset > 0; offset /= 2)
            tmp += __shfl_down_sync(mask, tmp, offset);
        if (threadIdx.x % warpSize == 0)
            cov[i] = tmp;
    }
}

__inline__ __device__ void reduce_sum_block(float* __restrict__ shared_mem, csize_t shared_chunks, csize_t cov_size)
{
    float* tmp_cov;
    unsigned mask = __ballot_sync(0xFFFFFFFF, threadIdx.x < shared_chunks);
    if (threadIdx.x < shared_chunks)
    {
        tmp_cov = shared_mem + cov_size * threadIdx.x;
        reduce_sum_warp(tmp_cov, cov_size, mask);
    }

    __syncthreads();

    auto ciel_div = (shared_chunks + warpSize - 1) / warpSize;

    mask = __ballot_sync(0xFFFFFFFF, threadIdx.x < ciel_div);
    if (threadIdx.x < ciel_div)
    {
        tmp_cov = shared_mem + cov_size * threadIdx.x * warpSize;
        reduce_sum_warp(tmp_cov, cov_size, mask);
    }
}

__inline__ __device__ void point_covariance(
    const float* __restrict__ point, csize_t dim, float* __restrict__ shared_mem)
{
    csize_t idx = 0;
    for (csize_t i = 0; i < dim; ++i)
        for (csize_t j = i; j < dim; ++j)
            atomicAdd(shared_mem + idx++, point[i] * point[j]);
}

__global__ void covariance(const float* __restrict__ points,
    csize_t dim,
    csize_t count,
    const asgn_t* __restrict__ assignments,
    float* __restrict__ cov_matrix,
    asgn_t cid,
    csize_t shared_chunks)
{
    csize_t cov_size = ((dim + 1) * dim) / 2;
    extern __shared__ float shared_mem[];
    float tmp_point[MAX_DIM];

    for (csize_t idx = threadIdx.x; idx < cov_size * shared_chunks; idx += blockDim.x)
        shared_mem[idx] = 0;

    __syncthreads();

    float* tmp_cov = shared_mem + cov_size * (threadIdx.x % shared_chunks);

    for (csize_t idx = blockDim.x * blockIdx.x + threadIdx.x; idx < count; idx += gridDim.x * blockDim.x)
        if (assignments[idx] == cid)
        {
            for (csize_t i = 0; i < dim; ++i)
                tmp_point[i] = points[idx * dim + i] - expected_point[i];

            point_covariance(tmp_point, dim, tmp_cov);
        }

    __syncthreads();

    reduce_sum_block(shared_mem, shared_chunks, cov_size);

    if (threadIdx.x == 0)
        for (csize_t i = 0; i < cov_size; ++i)
            atomicAdd(cov_matrix + i, tmp_cov[i]);
}

__global__ void finish_covariance(
    const float* __restrict__ in_cov_matrix, csize_t divisor, csize_t dim, float* __restrict__ out_cov_matrix)
{
    csize_t cov_size = ((dim + 1) * dim) / 2;

    for (csize_t idx = threadIdx.x; idx < cov_size; idx += blockDim.x)
    {
        auto coords = compute_coordinates(dim, idx);
        auto tmp = in_cov_matrix[idx] / divisor;
        out_cov_matrix[coords.x + coords.y * dim] = tmp;
        out_cov_matrix[coords.x * dim + coords.y] = tmp;
    }
}

__global__ void store_icov_data(float* __restrict__ icov_dest,
    float* __restrict__ mf_dest,
    const float* __restrict__ icov_src,
    const float mf_src,
    clustering::csize_t dim)
{
    csize_t cov_size = ((dim + 1) * dim) / 2;

    for (csize_t idx = threadIdx.x; idx < cov_size; idx += blockDim.x)
    {
        auto coords = compute_coordinates(dim, idx);

        if (coords.x == coords.y)
            icov_dest[idx] = icov_src[coords.x + coords.y * dim];
        else
            icov_dest[idx] = 2 * icov_src[coords.x + coords.y * dim];
    }

    if (threadIdx.x == 0 && mf_dest)
        *mf_dest = mf_src;
}

__device__ void reduce_mul_warp(float* __restrict__ point)
{
    for (unsigned int offset = warpSize / 2; offset > 0; offset /= 2)
        *point *= __shfl_down_sync(0xFFFFFFFF, *point, offset);
}

__device__ void reduce_mul_block(float* __restrict__ point, float* __restrict__ shared_mem)
{
    reduce_mul_warp(point);

    auto lane_id = threadIdx.x % warpSize;
    auto warp_id = threadIdx.x / warpSize;

    if (lane_id == 0)
        memcpy(shared_mem + warp_id, point, sizeof(float));

    __syncthreads();

    *point = (threadIdx.x < blockDim.x / warpSize) ? shared_mem[threadIdx.x] : 1;

    reduce_mul_warp(point);
}

__global__ void transform_cov(float* __restrict__ matrix,
    csize_t dim,
    float weight_factor,
    bool use_cholesky,
    const float* __restrict__ cholesky_decomp,
    const int* __restrict__ cholesky_success)
{
    __shared__ float shared[32];

    float mf = 1.f;

    if (use_cholesky && *cholesky_success == 0)
    {
        for (csize_t idx = threadIdx.x; idx < dim; idx += blockDim.x)
            mf *= powf(cholesky_decomp[idx * (dim + 1)], 2.f / dim);

        __syncthreads();

        reduce_mul_block(&mf, shared);

        if (threadIdx.x == 0)
            shared[0] = mf;

        __syncthreads();

        mf = shared[0];
    }

    csize_t cov_size = ((dim + 1) * dim) / 2;

    for (csize_t idx = threadIdx.x; idx < cov_size; idx += blockDim.x)
    {
        auto coords = compute_coordinates(dim, idx);

        matrix[coords.x + coords.y * dim] =
            weight_factor * matrix[coords.x + coords.y * dim] + (1 - weight_factor) * mf * (coords.x == coords.y);
    }
}

__global__ void compute_store_icov_mf(float* __restrict__ dest, csize_t dim, const float* __restrict__ cholesky_decomp)
{
    __shared__ float shared[32];

    float icmf = 1.f;

    for (csize_t idx = threadIdx.x; idx < dim; idx += blockDim.x)
        icmf *= powf(cholesky_decomp[idx * (dim + 1)], -2.f / dim);

    __syncthreads();

    reduce_mul_block(&icmf, shared);

    if (threadIdx.x == 0)
        *dest = icmf;
}


void run_covariance(const input_t in, const asgn_t* assignments, float* out, asgn_t centroid_id, kernel_info info)
{
    csize_t cov_size = ((in.dim + 1) * in.dim) / 2;
    csize_t shared_chunks = 10000 / cov_size;

    CUCH(hipMemset(out, 0, cov_size * sizeof(float)));
    CUCH(hipDeviceSynchronize());
    covariance<<<info.grid_dim, info.block_dim, shared_chunks * cov_size * sizeof(float)>>>(
        in.data, in.dim, in.count, assignments, out, centroid_id, shared_chunks);
}

void run_finish_covariance(const float* in_cov_matrix, csize_t divisor, csize_t dim, float* out_cov_matrix)
{
    finish_covariance<<<1, 32>>>(in_cov_matrix, divisor, dim, out_cov_matrix);
}

void run_store_icovariance_data(
    float* icov_dest, float* mf_dest, const float* icov_src, const float mf_src, clustering::csize_t dim)
{
    store_icov_data<<<1, 32>>>(icov_dest, mf_dest, icov_src, mf_src, dim);
}

void run_transform_cov(float* matrix,
    csize_t dim,
    float weight_factor,
    bool use_cholesky,
    const float* cholesky_decomp,
    const int* cholesky_success)
{
    transform_cov<<<1, 32>>>(matrix, dim, weight_factor, use_cholesky, cholesky_decomp, cholesky_success);
}

void run_compute_store_icov_mf(float* dest, csize_t dim, const float* cholesky_decomp)
{
    compute_store_icov_mf<<<1, 32>>>(dest, dim, cholesky_decomp);
}