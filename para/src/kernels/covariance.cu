#include "hip/hip_runtime.h"
#include <kernels.cuh>

#include <>

#include "common_kernels.cuh"

using namespace clustering;

__constant__ float expected_point[MAX_DIM];

void assign_constant_storage(const float* value, csize_t size, hipMemcpyKind kind)
{
	CUCH(hipMemcpyToSymbol(HIP_SYMBOL(expected_point), value, size, (size_t)0, kind));
}

__inline__ __device__ void reduce_sum_warp(float* __restrict__ cov, csize_t size, unsigned mask)
{
	for (csize_t i = 0; i < size; ++i)
	{
		float tmp = cov[i];
		for (unsigned int offset = warpSize / 2; offset > 0; offset /= 2)
			tmp += __shfl_down_sync(mask, tmp, offset);
		if (threadIdx.x % warpSize == 0)
			cov[i] = tmp;
	}
}

__inline__ __device__ void reduce_sum_block(float* __restrict__ shared_mem, csize_t shared_chunks, csize_t cov_size)
{
	float* tmp_cov;
	unsigned mask = __ballot_sync(0xFFFFFFFF, threadIdx.x < shared_chunks);
	if (threadIdx.x < shared_chunks)
	{
		tmp_cov = shared_mem + cov_size * threadIdx.x;
		reduce_sum_warp(tmp_cov, cov_size, mask);
	}

	__syncthreads();

	auto ciel_div = (shared_chunks + warpSize - 1) / warpSize;

	mask = __ballot_sync(0xFFFFFFFF, threadIdx.x < ciel_div);
	if (threadIdx.x < ciel_div)
	{
		tmp_cov = shared_mem + cov_size * threadIdx.x * warpSize;
		reduce_sum_warp(tmp_cov, cov_size, mask);
	}
}

__inline__ __device__ void point_covariance(const float* __restrict__ point, csize_t dim, float* __restrict__ shared_mem)
{
	csize_t idx = 0;
	for (csize_t i = 0; i < dim; ++i)
		for (csize_t j = i; j < dim; ++j)
			atomicAdd(shared_mem + idx++, point[i] * point[j]);
}

__global__ void covariance(const float* __restrict__ points, csize_t dim, csize_t count, const asgn_t* __restrict__ assignments, float* __restrict__ cov_matrix, asgn_t cid, csize_t shared_chunks)
{
	csize_t cov_size = ((dim + 1) * dim) / 2;
	extern __shared__ float shared_mem[];
	float tmp_point[MAX_DIM];

	for (csize_t idx = threadIdx.x; idx < cov_size * shared_chunks; idx += blockDim.x)
		shared_mem[idx] = 0;

	__syncthreads();

	float* tmp_cov = shared_mem + cov_size * (threadIdx.x % shared_chunks);

	for (csize_t idx = blockDim.x * blockIdx.x + threadIdx.x; idx < count; idx += gridDim.x * blockDim.x)
		if (assignments[idx] == cid)
		{
			for (csize_t i = 0; i < dim; ++i)
				tmp_point[i] = points[idx * dim + i] - expected_point[i];

			point_covariance(tmp_point, dim, tmp_cov);
		}

	__syncthreads();

	reduce_sum_block(shared_mem, shared_chunks, cov_size);

	if (threadIdx.x == 0)
		for (csize_t i = 0; i < cov_size; ++i)
			atomicAdd(cov_matrix + i, tmp_cov[i]);
}

__global__ void finish_covariance(const float* __restrict__ in_cov_matrix, csize_t divisor, csize_t dim, float* __restrict__ out_cov_matrix)
{
	csize_t cov_size = ((dim + 1) * dim) / 2;

	for (csize_t idx = threadIdx.x; idx < cov_size; idx+= blockDim.x)
	{
		auto coords = compute_coordinates(dim, idx);
		auto tmp = in_cov_matrix[idx] / divisor;
		out_cov_matrix[coords.x + coords.y * dim] = tmp;
		out_cov_matrix[coords.x * dim + coords.y] = tmp;
	}
}

__global__ void store_icov(float* __restrict__ dest, const float* __restrict__ src, asgn_t dim)
{
	csize_t cov_size = ((dim + 1) * dim) / 2;

	for (csize_t idx = threadIdx.x; idx < cov_size; idx += blockDim.x)
	{
		auto coords = compute_coordinates(dim, idx);
		
		if (coords.x == coords.y)
			dest[idx] = src[coords.x + coords.y * dim];
		else
			dest[idx] = 2 * src[coords.x + coords.y * dim];
	}
}

void run_covariance(const input_t in, const asgn_t* assignments, float* out, asgn_t centroid_id, kernel_info info)
{
	csize_t cov_size = ((in.dim + 1) * in.dim) / 2;
	csize_t shared_chunks = 10000 / cov_size;
	
	CUCH(hipMemset(out, 0, cov_size * sizeof(float)));
	CUCH(hipDeviceSynchronize());
	covariance << <info.grid_dim, info.block_dim, shared_chunks* cov_size * sizeof(float) >> > (in.data, in.dim, in.count, assignments, out, centroid_id, shared_chunks);
}

void run_finish_covariance(const float* in_cov_matrix, csize_t divisor, csize_t dim, float* out_cov_matrix)
{
	finish_covariance<<<1, ((dim + 1) * dim) / 2>>>(in_cov_matrix, divisor, dim, out_cov_matrix);
}

void run_store_icovariance(float* dest, const float* src, csize_t dim)
{
	store_icov<<<1, ((dim + 1) * dim) / 2 >>>(dest, src, dim);
}