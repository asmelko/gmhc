#include "hip/hip_runtime.h"
#include <kernels.cuh>

#include <>

using namespace clustering;

template <size_t N>
__device__ void add_neighbour(neighbour_t* neighbours, neighbour_t neighbour)
{
	neighbour_t prev_min;
	size_t i = 0;
	for (; i < N; ++i)
	{
		if (neighbours[i].distance > neighbour.distance)
		{
			prev_min = neighbours[i];
			neighbours[i] = neighbour;
			break;
		}
	}

	for (++i; i < N; i++)
	{
		if (prev_min.distance == FLT_MAX)
			return;

		neighbour_t tmp = neighbours[i];
		neighbours[i] = prev_min;
		prev_min = tmp;
	}
}

template <size_t N>
__device__ void merge_neighbours(const neighbour_t* l_neighbours, const neighbour_t* r_neighbours, neighbour_t* res)
{
	size_t l_idx = 0, r_idx = 0;

	for (size_t i = 0; i < N; ++i)
	{
		if (l_neighbours[l_idx].distance < r_neighbours[r_idx].distance)
			res[i] = l_neighbours[l_idx++];
		else
			res[i] = r_neighbours[r_idx++];
	}
}


__inline__ __device__ float euclidean_norm(const float* l_point, const float* r_point, size_t dim)
{
	float tmp_sum = 0;
	for (size_t i = 0; i < dim; ++i)
	{
		auto tmp = l_point[i] - r_point[i];
		tmp_sum += tmp * tmp;
	}
	return sqrtf(tmp_sum);
}

__inline__ __device__ void reduce_sum_warp(float* point, size_t dim)
{
	for (unsigned int offset = warpSize / 2; offset > 0; offset /= 2)
	{
		for (size_t i = 0; i < dim; ++i)
		{
			point[i] += __shfl_down_sync(0xFFFFFFFF, point[i], offset);
		}
	}
}


template <size_t N>
__inline__ __device__ void reduce_min_warp(neighbour_t* neighbours)
{
	for (unsigned int offset = warpSize / 2; offset > 0; offset /= 2)
	{
		neighbour_t tmp[N];
		for (size_t i = 0; i < N; ++i)
		{
			tmp[i].distance = __shfl_down_sync(0xFFFFFFFF, neighbours[i].distance, offset);
			tmp[i].idx = __shfl_down_sync(0xFFFFFFFF, neighbours[i].idx, offset);
		}

		neighbour_t tmp_cpy[N];
		merge_neighbours<N>(neighbours, tmp, tmp_cpy);
		memcpy(neighbours, tmp_cpy, sizeof(neighbour_t) * N);
	}
}

template <size_t N>
__inline__ __device__ void reduce_min_block(neighbour_t* neighbours, neighbour_t* shared_mem, bool reduce_warp = true)
{
	if (reduce_warp)
		reduce_min_warp<N>(neighbours);

	auto lane_id = threadIdx.x % warpSize;
	auto warp_id = threadIdx.x / warpSize;

	if (lane_id == 0)
		memcpy(shared_mem + warp_id * N, neighbours, sizeof(neighbour_t) * N);

	__syncthreads();

	if (threadIdx.x < blockDim.x / warpSize)
		memcpy(neighbours, shared_mem + threadIdx.x * N, sizeof(neighbour_t) * N);
	else
		for (size_t i = 0; i < N; i++)
			neighbours[i].distance = FLT_MAX;

	reduce_min_warp<N>(neighbours);
}

template <size_t N>
__inline__ __device__ void point_reduce(const neighbour_t* neighbours, size_t to_reduce, size_t count, neighbour_t* reduced, size_t idx)
{
	size_t block = idx / warpSize;
	neighbour_t local[N];

	size_t nei = idx % warpSize;

	if (nei < to_reduce)
		memcpy(local, neighbours + (block * to_reduce + nei) * N, sizeof(neighbour_t) * N);
	else
		for (size_t i = 0; i < N; i++)
			local[i].distance = FLT_MAX;


	for (nei += warpSize; nei < to_reduce; nei += warpSize)
	{
		neighbour_t tmp[N];
		merge_neighbours<N>(local, neighbours + (block * to_reduce + nei)*N, tmp);
		memcpy(local, tmp, sizeof(neighbour_t) * N);
	}


	reduce_min_warp<N>(local);


	if (threadIdx.x % warpSize == 0)
	{
		memcpy(reduced + block*N, local, sizeof(neighbour_t) * N);
	}
}


template <size_t N>
__global__ void reduce(const neighbour_t* neighbours, size_t to_reduce, size_t count, neighbour_t* reduced)
{
	for (size_t idx = threadIdx.x + blockIdx.x * blockDim.x; idx < count * warpSize; idx += blockDim.x * gridDim.x)
	{
		point_reduce<N>(neighbours, to_reduce, count, reduced, idx);
	}
}

template <size_t N>
__global__ void update_neighbours(size_t centroid_count, neighbour_t* neighbours_a, uint8_t* updated, size_t old_i, size_t old_j)
{
	extern __shared__ float shared_mem[];

	auto idx = threadIdx.x + blockDim.x * blockIdx.x;

	for (asgn_t x = idx; x < centroid_count; x += blockDim.x * gridDim.x)
	{
		if (x == old_i || x == old_j)
		{
			updated[x] = 1;
			continue;
		}

		neighbour_t tmp_nei[N];
		memcpy(tmp_nei, neighbours_a + x * N, sizeof(neighbour_t) * N);

		size_t last_empty = 0;

		for (size_t i = 0; i < N; i++)
		{
			if (tmp_nei[i].distance == FLT_MAX)
				break;

			if (tmp_nei[i].idx == old_i || tmp_nei[i].idx == old_j)
				tmp_nei[i].distance = FLT_MAX;
			else
			{
				if (tmp_nei[i].idx == centroid_count)
					tmp_nei[i].idx = old_j;

				tmp_nei[last_empty++] = tmp_nei[i];
			}
		}

		updated[x] = tmp_nei[0].distance == FLT_MAX ? 1 : 0;

		memcpy(neighbours_a + x * N,tmp_nei, sizeof(neighbour_t) * N);
	}
}

template <size_t N>
__global__ void reduce_u(const neighbour_t* neighbours, size_t to_reduce, size_t count, neighbour_t* reduced, uint8_t* updated)
{
	for (size_t idx = threadIdx.x + blockIdx.x * blockDim.x; idx < count * warpSize; idx += blockDim.x * gridDim.x)
	{
		if (updated[idx / warpSize])
			point_reduce<N>(neighbours, to_reduce, count, reduced, idx);
	}
}

__inline__ __device__ chunk_t reduce_min_warp(chunk_t data)
{
	for (unsigned int offset = warpSize / 2; offset > 0; offset /= 2)
	{
		auto tmp_dist = __shfl_down_sync(0xFFFFFFFF, data.min_dist, offset);
		auto tmp_i = __shfl_down_sync(0xFFFFFFFF, data.min_i, offset);
		auto tmp_j = __shfl_down_sync(0xFFFFFFFF, data.min_j, offset);
		if (tmp_dist < data.min_dist)
		{
			data.min_dist = tmp_dist;
			data.min_i = tmp_i;
			data.min_j = tmp_j;
		}
	}
	return data;
}

__inline__ __device__ chunk_t reduce_min_block(chunk_t data, chunk_t* shared_mem)
{
	data = reduce_min_warp(data);

	auto lane_id = threadIdx.x % warpSize;
	auto warp_id = threadIdx.x / warpSize;

	if (lane_id == 0)
		shared_mem[warp_id] = data;

	__syncthreads();

	data = (threadIdx.x < blockDim.x / warpSize) ? shared_mem[threadIdx.x] : shared_mem[0];

	data = reduce_min_warp(data);
	return data;
}

template <size_t N>
__global__ void min(const neighbour_t* neighbours, size_t count, chunk_t* result)
{
	static __shared__ chunk_t shared_mem[32];

	chunk_t tmp;
	tmp.min_dist = FLT_MAX;
	for (size_t idx = threadIdx.x; idx < count; idx += blockDim.x)
	{
		if (tmp.min_dist > neighbours[idx*N].distance)
		{
			tmp.min_dist = neighbours[idx * N].distance;
			tmp.min_j = neighbours[idx * N].idx;
			tmp.min_i = idx;
		}
	}

	tmp = reduce_min_block(tmp, shared_mem);

	if (threadIdx.x == 0)
		*result = tmp;
}

__global__ void print_up(uint8_t* updated, size_t count)
{
	for (size_t i = 0; i < count; i++)
	{
		printf("%d. %d\n", (int)i, updated[i]);
	}
}

__global__ void print_ne(neighbour_t* neighbours, size_t nei_number, size_t count)
{
	for (size_t i = 0; i < count; i++)
	{
		printf("%d. %f %d\n", (int)i, neighbours[i * nei_number].distance, (int)neighbours[i* nei_number].idx);
	}
}

void print_nei(neighbour_t* neighbours, size_t nei_number, size_t count)
{
	print_ne << <1, 1 >> > (neighbours, nei_number, count);
}

#include "neighbour_eucl.cu"
#include "neighbour_maha.cu"

template <size_t N>
void run_update_neighbours(const float* centroids, const float*const* inverses, size_t dim, size_t centroid_count, neighbour_t* tmp_neighbours, neighbour_t* act_neighbours, cluster_kind* cluster_kinds, uint8_t* updated, size_t old_i, size_t old_j, kernel_info info)
{
	size_t shared = dim * sizeof(float) + 32 * sizeof(neighbour_t) * N;
	size_t shared_mat = (dim + 33) * dim * sizeof(float) + 32 * sizeof(neighbour_t) * N;
	update_neighbours<N><<<info.grid_dim, info.block_dim >>>(centroid_count, act_neighbours, updated, old_i, old_j);

	neighbours_u<N> << <info.grid_dim, info.block_dim, shared >> > (centroids, dim, centroid_count, tmp_neighbours, cluster_kinds, updated, old_i);
	neighbours_mat_u<N> << <info.grid_dim, info.block_dim, shared_mat >> > (centroids, inverses, dim, centroid_count, tmp_neighbours, cluster_kinds, updated, old_i);

	CUCH(hipGetLastError());
	CUCH(hipDeviceSynchronize());

	reduce_u<N><<<info.grid_dim, info.block_dim>>>(tmp_neighbours, info.grid_dim, centroid_count, act_neighbours, updated);
}

template <size_t N>
void run_neighbours(const float* centroids, size_t dim, size_t centroid_count, neighbour_t* tmp_neighbours, neighbour_t* act_neighbours, cluster_kind* cluster_kinds, kernel_info info)
{
	size_t shared = dim * sizeof(float) + 32 * sizeof(neighbour_t) * N;
	neighbours<N> << <info.grid_dim, info.block_dim, shared >> > (centroids, dim, centroid_count, tmp_neighbours, cluster_kinds);
	reduce<N> << <info.grid_dim, info.block_dim >> > (tmp_neighbours, info.grid_dim, centroid_count, act_neighbours);
}

template <size_t N>
chunk_t run_neighbours_min(const neighbour_t* neighbours, size_t count, chunk_t* result)
{
	min<N> << <1, 1024 >> > (neighbours, count, result);

	CUCH(hipDeviceSynchronize());

	chunk_t res;
	CUCH(hipMemcpy(&res, result, sizeof(chunk_t), hipMemcpyKind::hipMemcpyDeviceToHost));

	if (res.min_i > res.min_j)
		std::swap(res.min_i, res.min_j);

	return res;
}

template void run_neighbours<1>(const float* centroids, size_t dim, size_t centroid_count, neighbour_t* tmp_neighbours, neighbour_t* neighbours, cluster_kind* cluster_kinds, kernel_info info);
template void run_neighbours<2>(const float* centroids, size_t dim, size_t centroid_count, neighbour_t* tmp_neighbours, neighbour_t* neighbours, cluster_kind* cluster_kinds, kernel_info info);
template void run_neighbours<5>(const float* centroids, size_t dim, size_t centroid_count, neighbour_t* tmp_neighbours, neighbour_t* neighbours, cluster_kind* cluster_kinds, kernel_info info);
template chunk_t run_neighbours_min<1>(const neighbour_t* neighbours, size_t count, chunk_t* result);
template chunk_t run_neighbours_min<2>(const neighbour_t* neighbours, size_t count, chunk_t* result);
template chunk_t run_neighbours_min<5>(const neighbour_t* neighbours, size_t count, chunk_t* result);
template void run_update_neighbours<1>(const float* centroids, const float* const* inverses, size_t dim, size_t centroid_count, neighbour_t* tmp_neighbours, neighbour_t* act_neighbours, cluster_kind* cluster_kinds, uint8_t* updated, size_t old_i, size_t old_j, kernel_info info);
template void run_update_neighbours<2>(const float* centroids, const float* const* inverses, size_t dim, size_t centroid_count, neighbour_t* tmp_neighbours, neighbour_t* act_neighbours, cluster_kind* cluster_kinds, uint8_t* updated, size_t old_i, size_t old_j, kernel_info info);
template void run_update_neighbours<5>(const float* centroids, const float* const* inverses, size_t dim, size_t centroid_count, neighbour_t* tmp_neighbours, neighbour_t* act_neighbours, cluster_kind* cluster_kinds, uint8_t* updated, size_t old_i, size_t old_j, kernel_info info);
