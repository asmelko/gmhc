#include "hip/hip_runtime.h"
#include <kernels.cuh>

#include <>
#include "common_kernels.cuh"

using namespace clustering;

template <size_t N>
__device__ void point_neighbour(const float* centroids, size_t dim, size_t centroid_count, neighbour_t* neighbours_a, float* shared_mem, cluster_kind* cluster_kinds, size_t idx, bool from_start)
{
	neighbour_t local_neighbours[N];

	for (size_t i = 0; i < N; ++i)
		local_neighbours[i].distance = FLT_MAX;

	for (size_t i = threadIdx.x; i < dim; i += blockDim.x)
		shared_mem[i] = centroids[idx * dim + i];

	__syncthreads();

	asgn_t y = threadIdx.x + blockIdx.x * blockDim.x;

	if (from_start)
		for (; y < idx; y += blockDim.x * gridDim.x)
		{
			if (cluster_kinds[y] != cluster_kind::EUCL)
				continue;

			//printf("from start %d\n", (int)y);

			float dist = euclidean_norm(shared_mem, centroids + y * dim, dim);

			add_neighbour<N>(local_neighbours, neighbour_t{ dist, y });
		}
	else
		y += idx;

	for (; y < centroid_count - 1; y += blockDim.x * gridDim.x)
	{
		if (cluster_kinds[y + 1] != cluster_kind::EUCL)
			continue;

		//printf("cycle %d %d\n", (int)y, (int)new_y);


		float dist = euclidean_norm(shared_mem, centroids + (y + 1) * dim, dim);

		add_neighbour<N>(local_neighbours, neighbour_t{ dist, y + 1 });
	}

	reduce_min_block<N>(local_neighbours, reinterpret_cast<neighbour_t*>(shared_mem + dim));

	if (threadIdx.x == 0)
	{
		memcpy(neighbours_a + (gridDim.x * idx + blockIdx.x) * N, local_neighbours, N * sizeof(neighbour_t));
	}
}

template <size_t N>
__global__ void neighbours(const float* centroids, size_t dim, size_t centroid_count, neighbour_t* neighbours_a, cluster_kind* cluster_kinds)
{
	extern __shared__ float shared_mem[];

	for (asgn_t x = 0; x < centroid_count; ++x)
	{
		if (cluster_kinds[x] != cluster_kind::EUCL)
			continue;

		point_neighbour<N>(centroids, dim, centroid_count, neighbours_a, shared_mem, cluster_kinds, x, false);
	}
}

template <size_t N>
__global__ void neighbours_u(const float* centroids, size_t dim, size_t centroid_count, neighbour_t* neighbours_a, cluster_kind* cluster_kinds, uint8_t* updated, size_t new_idx)
{
	extern __shared__ float shared_mem[];

	for (asgn_t x = 0; x < centroid_count; ++x)
	{
		if (!updated[x] || cluster_kinds[x] != cluster_kind::EUCL)
			continue;

		point_neighbour<N>(centroids, dim, centroid_count, neighbours_a, shared_mem, cluster_kinds, x, x == new_idx);
	}
}